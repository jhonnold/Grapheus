#include "hip/hip_runtime.h"

#include "chess/chess.h"
#include "dataset/batchloader.h"
#include "dataset/dataset.h"
#include "dataset/io.h"
#include "dataset/process.h"
#include "misc/csv.h"
#include "misc/timer.h"
#include "nn/nn.h"
#include "operations/operations.h"

#include <fstream>

using namespace nn;
using namespace data;

struct ChessModel : nn::Model {

    // seting inputs
    virtual void setup_inputs_and_outputs(dataset::DataSet<chess::Position>* positions,
                                          const float                        lambda = 1.0) = 0;

    // train function
    void train(dataset::BatchLoader<chess::Position>& loader,
               dataset::BatchLoader<chess::Position>& validation_loader,
               int                                    epochs          = 1500,
               int                                    epoch_size      = 1e8,
               int                                    validation_size = 1e7) {
        this->compile(loader.batch_size);

        Timer t {};
        for (int i = 1; i <= epochs; i++) {
            t.start();

            uint64_t prev_print_tm         = 0;
            float    total_epoch_loss      = 0;
            float    total_validation_loss = 0;

            for (int b = 1; b <= epoch_size / loader.batch_size; b++) {
                auto* ds = loader.next();
                setup_inputs_and_outputs(ds, 0.5);

                float batch_loss = batch();
                total_epoch_loss += batch_loss;
                float epoch_loss = total_epoch_loss / b;

                t.stop();
                uint64_t elapsed = t.elapsed();
                if (elapsed - prev_print_tm > 1000 || b == epoch_size / loader.batch_size) {
                    prev_print_tm = elapsed;

                    printf("\rep = [%4d], epoch_loss = [%1.8f], batch = [%5d], batch_loss = [%1.8f], "
                           "speed = [%7d pos/s], time = [%3ds]",
                           i,
                           epoch_loss,
                           b,
                           batch_loss,
                           (int) (1000.0f * loader.batch_size * b / elapsed),
                           (int) (elapsed / 1000.0f));
                    std::cout << std::flush;
                }
            }

            std::cout << std::endl;

            float epoch_loss = total_epoch_loss / (epoch_size / loader.batch_size);

            for (int b = 1; b <= validation_size / validation_loader.batch_size; b++) {
                auto* ds = validation_loader.next();
                setup_inputs_and_outputs(ds, 0.5);

                total_validation_loss += loss();
            }

            float validation_loss =
                total_validation_loss / (validation_size / validation_loader.batch_size);
            printf("ep = [%4d], valid_loss = [%1.8f]", i, validation_loss);
            std::cout << std::endl;

            next_epoch(epoch_loss, validation_loss);

            // distribution(validation_loader);
        }
    }

    void test_fen(const std::string& fen) {
        this->compile(1);

        chess::Position                   pos = chess::parse_fen(fen);
        dataset::DataSet<chess::Position> ds {};
        ds.positions.push_back(pos);
        ds.header.entry_count = 1;

        // setup inputs of network
        setup_inputs_and_outputs(&ds);

        // forward pass
        this->upload_inputs();
        this->forward();

        // go through the layers and download values

        std::cout
            << "==================================================================================\n";
        std::cout << "testing fen: " << fen << std::endl;

        int idx = 0;
        for (auto layer : m_layers) {
            layer->dense_output.values >> CPU;

            std::cout << "LAYER " << ++idx << std::endl;
            for (int i = 0; i < std::min((size_t) 16, layer->size); i++) {
                std::cout << std::setw(10) << layer->dense_output.values(i, 0);
            }
            if (layer->size > 16) {
                std::cout << " ......... " << layer->dense_output.values(layer->size - 1, 0);
            }
            std::cout << "\n";
        }
    }

    void distribution(dataset::BatchLoader<chess::Position>& loader, int batches = 32) {
        this->compile(loader.batch_size);

        std::vector<DenseMatrix<float>> max_values {};
        std::vector<DenseMatrix<float>> min_values {};

        for (auto l : m_layers) {
            max_values.emplace_back(l->dense_output.values.m, 1);
            min_values.emplace_back(l->dense_output.values.m, 1);
            max_values.back().malloc<data::CPU>();
            min_values.back().malloc<data::CPU>();
            math::uniform(max_values.back(), -1000000.0f, -1000000.0f);
            math::uniform(min_values.back(), 1000000.0f, 1000000.0f);
        }

        for (int b = 0; b < batches; b++) {
            auto* ds = loader.next();
            setup_inputs_and_outputs(ds);
            this->upload_inputs();
            this->forward();
            std::cout << "\r" << b << " / " << batches << std::flush;

            // get minimum and maximum values
            for (int i = 0; i < m_layers.size(); i++) {
                auto layer = m_layers[i].get();
                layer->dense_output.values >> data::CPU;
                for (int m = 0; m < layer->dense_output.values.m; m++) {
                    for (int n = 0; n < layer->dense_output.values.n; n++) {
                        max_values[i](m, 0) =
                            std::max(max_values[i](m, 0), layer->dense_output.values(m, n));
                        min_values[i](m, 0) =
                            std::min(min_values[i](m, 0), layer->dense_output.values(m, n));
                    }
                }
            }
        }
        std::cout << std::endl;

        for (int i = 0; i < m_layers.size(); i++) {
            std::cout << "------------ LAYER " << i + 1 << " --------------------" << std::endl;
            std::cout << "min: ";
            for (int j = 0; j < std::min((size_t) 16, min_values[i].size()); j++) {
                std::cout << std::setw(10) << min_values[i](j);
            }
            if (min_values[i].size() > 16) {
                std::cout << " ......... " << min_values[i](min_values.size() - 1);
            }
            std::cout << "\n";

            std::cout << "max: ";
            for (int j = 0; j < std::min((size_t) 16, max_values[i].size()); j++) {
                std::cout << std::setw(10) << max_values[i](j);
            }
            if (max_values[i].size() > 16) {
                std::cout << " ......... " << max_values[i](max_values.size() - 1);
            }

            std::cout << "\n";
            float min = 10000000;
            float max = -10000000;
            for (int m = 0; m < min_values.size(); m++) {
                min = std::min(min, min_values[i](m));
                max = std::max(max, max_values[i](m));
            }
            std::cout << "output bounds: [" << min << " ; " << max << "]\n";

            int died = 0;
            for (int j = 0; j < max_values[i].size(); j++) {
                if (std::abs(max_values[i](j) - min_values[i](j)) < 1e-8) {
                    died++;
                }
            }

            std::cout << "died: " << died << " / " << max_values[i].size();
            std::cout << "\n";

            for (auto p : m_layers[i]->params()) {
                float min = 10000000;
                float max = -10000000;
                for (int m = 0; m < p->values.m; m++) {
                    for (int n = 0; n < p->values.n; n++) {
                        min = std::min(min, p->values(m, n));
                        max = std::max(max, p->values(m, n));
                    }
                }

                std::cout << "param bounds: [" << min << " ; " << max << "]\n";
            }
        }
    }
};

struct BerserkModel : ChessModel {
    SparseInput* in1;
    SparseInput* in2;
    DenseInput*  layer_sel;

    const float  sigmoid_scale = 1.0 / 160.0;
    const float  quant_one     = 64.0;
    const float  quant_two     = 32.0;

    const size_t n_features    = 16 * 12 * 64;
    const size_t n_ft          = 768;
    const size_t n_l1          = 8;
    const size_t n_l2          = 32;
    const size_t n_out         = 1;
    const size_t n_layers      = 8;

    BerserkModel()
        : ChessModel() {

        in1                   = add<SparseInput>(n_features, 32);
        in2                   = add<SparseInput>(n_features, 32);
        layer_sel             = add<DenseInput>(1);

        auto ft               = add<FeatureTransformer>(in1, in2, n_ft);
        auto fta              = add<ReLU>(ft);
        ft->ft_regularization = 1.0 / 16384.0 / 4194304.0;

        auto l1               = add<AffineMulti>(fta, n_l1, n_layers);
        auto l1a              = add<ReLU>(l1);

        auto l2               = add<AffineBatched>(l1a, n_l2 * n_layers, n_layers);
        auto l2a              = add<ReLU>(l2);

        auto pos_eval         = add<AffineBatched>(l2a, n_out * n_layers, n_layers);
        auto layer_eval       = add<SelectSingle>(pos_eval, layer_sel, n_layers);
        auto sigmoid          = add<Sigmoid>(layer_eval, sigmoid_scale);

        // Mean power error
        set_loss(MPE {2.5, true});

        // Steady LR decay
        set_lr_schedule(StepDecayLRSchedule {5e-3, 0.025, 1000});

        const float hidden_max = 127.0 / quant_two;

        add_optimizer(Adam({{OptimizerEntry {&ft->weights}},
                            {OptimizerEntry {&ft->bias}},
                            {OptimizerEntry {&l1->weights}.clamp(-hidden_max, hidden_max)},
                            {OptimizerEntry {&l1->bias}},
                            {OptimizerEntry {&l2->weights}},
                            {OptimizerEntry {&l2->bias}},
                            {OptimizerEntry {&pos_eval->weights}},
                            {OptimizerEntry {&pos_eval->bias}}},
                           0.95,
                           0.999,
                           1e-8));

        set_file_output("C:/Programming/berserk-nets/exp15/");

        add_quantization(Quantizer {
            "" + std::to_string((int) quant_one) + "_" + std::to_string((int) quant_two),
            10,
            QuantizerEntry<int16_t>(&ft->weights.values, quant_one, true),
            QuantizerEntry<int16_t>(&ft->bias.values, quant_one),
            QuantizerEntry<int8_t>(&l1->weights.values, quant_two),
            QuantizerEntry<int32_t>(&l1->bias.values, quant_two),
            QuantizerEntry<float>(&l2->weights.values, 1.0),
            QuantizerEntry<float>(&l2->bias.values, quant_two),
            QuantizerEntry<float>(&pos_eval->weights.values, 1.0),
            QuantizerEntry<float>(&pos_eval->bias.values, quant_two),
        });
        set_save_frequency(10);
    }

    inline int king_square_index(int relative_king_square) {
        constexpr int indices[64] {
            -1, -1, -1, -1, 14, 14, 15, 15,    //
            -1, -1, -1, -1, 14, 14, 15, 15,    //
            -1, -1, -1, -1, 12, 12, 13, 13,    //
            -1, -1, -1, -1, 12, 12, 13, 13,    //
            -1, -1, -1, -1, 8,  9,  10, 11,    //
            -1, -1, -1, -1, 8,  9,  10, 11,    //
            -1, -1, -1, -1, 4,  5,  6,  7,     //
            -1, -1, -1, -1, 0,  1,  2,  3,     //
        };

        return indices[relative_king_square];
    }

    inline int index(chess::Square piece_square,
                     chess::Piece  piece,
                     chess::Square king_square,
                     chess::Color  view) {

        const chess::PieceType piece_type  = chess::type_of(piece);
        const chess::Color     piece_color = chess::color_of(piece);

        piece_square ^= 56;
        king_square ^= 56;

        chess::Square relative_king_square;
        chess::Square relative_piece_square;

        const int     oP  = piece_type + 6 * (piece_color != view);
        const int     oK  = (7 * !(king_square & 4)) ^ (56 * view) ^ king_square;
        const int     oSq = (7 * !(king_square & 4)) ^ (56 * view) ^ piece_square;

        return king_square_index(oK) * 12 * 64 + oP * 64 + oSq;
    }

    void setup_inputs_and_outputs(dataset::DataSet<chess::Position>* positions, const float lambda) {
        in1->sparse_output.clear();
        in2->sparse_output.clear();

        auto& target = m_loss->target;

#pragma omp parallel for schedule(static) num_threads(16)
        for (int b = 0; b < positions->header.entry_count; b++) {
            chess::Position* pos = &positions->positions[b];
            // fill in the inputs and target values

            chess::Square wKingSq = pos->get_king_square<chess::WHITE>();
            chess::Square bKingSq = pos->get_king_square<chess::BLACK>();

            chess::BB     bb {pos->m_occupancy};
            int           idx   = 0;

            while (bb) {
                chess::Square sq = chess::lsb(bb);
                chess::Piece  pc = pos->m_pieces.get_piece(idx);

                auto piece_index_white_pov = index(sq, pc, wKingSq, chess::WHITE);
                auto piece_index_black_pov = index(sq, pc, bKingSq, chess::BLACK);

                if (pos->m_meta.stm() == chess::WHITE) {
                    in1->sparse_output.set(b, piece_index_white_pov);
                    in2->sparse_output.set(b, piece_index_black_pov);
                } else {
                    in2->sparse_output.set(b, piece_index_white_pov);
                    in1->sparse_output.set(b, piece_index_black_pov);
                }

                bb = chess::lsb_reset(bb);
                idx++;
            }

            float p_value = pos->m_result.score;
            float w_value = pos->m_result.wdl;

            // flip if black is to move -> relative network style
            if (pos->m_meta.stm() == chess::BLACK) {
                p_value = -p_value;
                w_value = -w_value;
            }

            float p_target                       = 1 / (1 + expf(-p_value * sigmoid_scale));
            float w_target                       = (w_value + 1) / 2.0f;

            target(b)                            = lambda * p_target + (1.0 - lambda) * w_target;

            int layer = (chess::popcount(pos->m_occupancy) - 1) / 4;
            layer_sel->dense_output.values(b, 0) = layer;
        }
    }
};

int main() {
    math::seed(0);

    init();

    std::vector<std::string> files {};
    for (int i = 1; i <= 200; i++)
        files.push_back("C:/Programming/berserk-data/exp203/exp203." + std::to_string(i) + ".bin");

    std::vector<std::string> validation_files {};
    validation_files.push_back("C:/Programming/berserk-data/exp203/validation.bin");

    const int                             batch_size = 16384;
    dataset::BatchLoader<chess::Position> loader {files, batch_size};
    loader.start();
    dataset::BatchLoader<chess::Position> validation_loader {validation_files, batch_size};
    validation_loader.start();

    BerserkModel model {};
    model.train(loader, validation_loader);

    loader.kill();
    validation_loader.kill();

    close();
    return 0;
}
